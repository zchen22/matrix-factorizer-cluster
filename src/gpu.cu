#include "gpu.h"

Gpu::Gpu(const int id, MpiInfo* mpi_info, Logger* logger)
    : id_(id), mpi_info_(mpi_info), logger_(logger) {
  hipError_t e = hipSuccess;
  e = hipStreamCreate(&shader_stream_);
  logger_->CheckCudaError(e);
  e = hipStreamCreate(&h2d_stream_);
  logger_->CheckCudaError(e);
  e = hipStreamCreate(&d2h_stream_);
  logger_->CheckCudaError(e);
  e = hipMemGetInfo(&free_global_mem_size_, &global_mem_size_);
  logger->CheckCudaError(e);
}

Gpu::~Gpu() {
}

